#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "RipplingMath.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    RipplingMath ripplingMath = RipplingMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i, j;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	ripplingMath.colorIJ(&ptrDevPixels[s], i, j, t);

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

