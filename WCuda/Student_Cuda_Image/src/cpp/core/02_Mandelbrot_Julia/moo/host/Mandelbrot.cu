#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot::Mandelbrot(uint nMin, uint nMax, const Grid& grid, uint w, uint h, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda_RGBA_uchar4", domaineMath), variateurAnimation(Interval<float>(nMin, nMax), 1.f)
    {
    // Tools
    this->t = nMin; // protected dans Animable
    }

Mandelbrot::~Mandelbrot()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    t = variateurAnimation.get();
    mandelbrot <<<dg,db>>>(ptrDevPixels,w,h,t,domaineMath);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
    {
    t = variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

