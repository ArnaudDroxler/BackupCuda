#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

#include "math/JuliaMath.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void julia(float c1, float c2, uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void julia(float c1, float c2, uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath)
    {
    JuliaMath juliaMath = JuliaMath(t, c1, c2);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i, j;
    double x, y;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	domaineMath.toXY(i, j, &x, &y);

	juliaMath.colorXY(&ptrDevPixels[s], x, y, t);

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

