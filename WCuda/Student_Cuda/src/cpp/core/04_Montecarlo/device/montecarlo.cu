#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "reductionADDLock.h"
#include <hiprand/hiprand_kernel.h>


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarlo(hiprandState* tabDevGeneratorGM, long nbDarts, long* ptrDevNxTotal);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x);
__device__ int mutex=0;
/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void montecarlo(hiprandState* tabDevGeneratorGM, long nbDarts, uint m, long* ptrDevNxTotal)
    {
    extern __shared__ long TAB_SM[];

    //reductionIntraThread(TAB_SM, nbSlice);
    // lancer les flechettes
    // Regarder si elle est en dessous
    // ADD myNx


    const int NB_THREAD=Indice1D::nbThread();
    const int TID=Indice1D::tid();
    const int TIDLocal = Indice1D::tidLocal();

    hiprandState generator = tabDevGeneratorGM[TID];

    long localNx = 0;
    float x;
    float y;

    for(long i = 0; i < nbDarts; i++)
	{
	x = hiprand_uniform(&generator);
	// * M NORMALEMENT VVV
	y = hiprand_uniform(&generator) * m;

	if ( y  < f(x) )
	    {
	    localNx ++;
	    }
	}

    TAB_SM[TIDLocal] = localNx;

    __syncthreads();

    Lock l = Lock(&mutex);
    reductionADD<long>(TAB_SM, ptrDevNxTotal, &l);
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return 4/(1+x*x);
    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

