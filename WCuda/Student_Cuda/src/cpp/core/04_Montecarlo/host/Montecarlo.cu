#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"
#include "Montecarlo.h"
#include <hiprand/hiprand_kernel.h>

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId);
extern __global__ void montecarlo(hiprandState* tabDevGeneratorGM, uint nbDarts, uint m, uint* ptrDevNxTotal);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid, double* ptrResult, int nbDartsTotal, float m)
    {
    this->ptrResult = ptrResult;
    this->nbDartsTotal = nbDartsTotal;
    this->m = m;
    this->sizeOctetResultGM = sizeof(uint); // octet
    this->sizeOctetGeneratorsGM = Device::nbThread(grid) * sizeof(hiprandState);
    this->sizeOctetSM = Device::nbThread(grid) * sizeof(uint);

    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevResult, sizeOctetResultGM);
	    Device::memclear(ptrDevResult, sizeOctetResultGM);

	    Device::malloc(&ptrDevGenerators, sizeOctetGeneratorsGM);
	    Device::memclear(ptrDevGenerators, sizeOctetGeneratorsGM);
	    }
	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}

    // Generators creation
    createGenerator<<<dg,db>>>(ptrDevGenerators, Device::getDeviceId()); // assynchrone

    // Calculate the number of darts for each threads
    this->nbDarts = nbDartsTotal / Device::nbThread(grid);
    }

Montecarlo::~Montecarlo(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevResult);
	Device::free(ptrDevGenerators);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Montecarlo::run()
    {
    Device::lastCudaError("montecarlo (before)"); // temp debug
    montecarlo<<<dg,db, sizeOctetSM>>>(ptrDevGenerators, nbDarts, m, ptrDevResult); // assynchrone
    Device::lastCudaError("montecarlo (after)"); // temp debug

    Device::synchronize(); // Temp,debug, only for printf in  GPU

    uint montecarloResult;

    // MM (Device -> Host)
	{
	Device::memcpyDToH(&montecarloResult, ptrDevResult, sizeOctetResultGM); // barriere synchronisation implicite
	}

    *ptrResult = (double)montecarloResult / (double) nbDartsTotal * m;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
